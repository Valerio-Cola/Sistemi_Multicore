
#include <hip/hip_runtime.h>
/*
Per ogni pixel allocato in una matrice 2D: red*0.21 + green*0.72 + blue*0.07

La matrice può essere vista come un array 1D composto dalle righe della matrice messe in sequenza

Accesso a singolo pixel [riga*len_riga+colonna]

L'immagine è divisa in blocchi di thread, 1 thread = 1 pixel
Può essere che i blocchi abbiano righe/colonne di pixel che non devono lavorare
    poichè la dimensione dell'immagine non è precisa alla griglia di blocchi 
*/

//Conversione immagine in bianco e nero
__global__ void colorToGrey(unsigned char *imageColor, unsigned char *imageGrey, int width, int height) {
    // Calcola la posizione globale del pixel 
    int col = blockIdx.x + threadIdx.x * blockDim.x;
    int row = blockIdx.y + threadIdx.y * blockDim.y;

    // Controlla che il pixel sia all'interno dell'immagine
    if(col < width && row < height) {
        // Calcola l'indice del pixel da scrivere
        int index = row * width + col;

        // Moltiplico per il numero di canali di ingresso RGB
        // così da ottenere l'indice del pixel nell'immagine a colori
        int colorIndex = index * 3;

        // Legge i valori dei canali di colore
        unsigned char r = imageColor[colorIndex];
        unsigned char g = imageColor[colorIndex + 1];
        unsigned char b = imageColor[colorIndex + 2];

        // Calcola il valore del pixel in scala di grigio
        imageGrey[index] = (0.21f*r + 0.71f*g + 0.07f*b);
    }
}
#define BLUR_SIZE 3

// Effetto blur sull'immagine, si ottiene applicando al pixel il valore medio di tutti i suoi pixel adiacenti
// Non del tutto corretto poichè non considera i pixel ai bordi di ogni blocco
// Pongo immagine con 1 canale
__global__ void blurImage(unsigned char *imageIn, unsigned char *imageOut, int width, int height) {
    
    // Calcola la posizione globale del pixel
    int col = blockIdx.x + threadIdx.x * blockDim.x;
    int row = blockIdx.y + threadIdx.y * blockDim.y;

    // Controlla che il pixel sia all'interno dell'immagine
    if(col < width && row < height) {
        int pixVal = 0;
        int pixels = 0;

        // Calcola la media dei pixel vicini quindi gli 8 pixel adiacenti per ottenere effetto blur
        for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
            for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
                
                // Indice di ogni pixel vicino
                int curRow = row + blurRow;
                int curCol = col + blurCol;

                // Controlla che il pixel sia all'interno dell'immagine
                if(curRow > -1 && curRow < height && curCol > -1 && curCol < width) {
                    pixVal += imageIn[curRow * width + curCol];
                    pixels++;
                }
            }
        }
    // Applica il valore medio calcolato diviso il numero di pixel vicini
    imageOut[row * width + col] = (unsigned char)(pixVal / pixels);
    }
}
