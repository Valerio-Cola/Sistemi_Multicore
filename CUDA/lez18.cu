#include <hip/hip_runtime.h>
#include <stdio.h>

#define RADIUS 3
#define BLOCK_SIZE 256

/*
    Problema, i thread lavorano in parallelo, in caso ci fossero più di 32 elementi vuol dire che verranno utilizzti più warp (ognuno 32)
    e quindi gli elementi caricati e calcolati dal primo potrebbero non essere soncronizzati con gli altri warp
    Soluzione: __syncthread() è una barriera che verifica che tutti i thread del blocco abbiano completato il calcolo prima di continuare
*/
__global__ void stencil_1D(int *in, int *out) {
    // crea un array in memoria condivisa per memorizzare i dati del blocco
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    
    // Calcola l'indice globale e locale
    int gindex = blockDim.x * blockIdx.x + threadIdx.x;
    int lindex = threadIdx.x + RADIUS;
    
    // Legge gil elementi in memoria globale e li copia in memoria condivisa
    temp[lindex] = in[gindex];
    
    if (threadIdx.x < RADIUS) {
        // I primi 3 elementi e gli ultimi 3 devono caricare in memoria condivisa anche elementi della regione di halo
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }
    
    // Sincronizza i thread del blocco in modo che tutti abbiano copiato i dati in memoria condivisa prima di continuare
    __syncthreads();

    // Ogni thread calcola la somma dei 7 elementi (3 a sinistra, 3 a destra e se stesso)
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++)
        result += temp[lindex + offset];

    // Scrive il risultato in memoria globale
    out[gindex] = result;
}

int main(int argc, char const *argv[])
{
    return 0;
}
